#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

__global__ void kernel_vecDouble(int *in, int *out, const size_t n) {
	int i = threadIdx.x;
	if (i < n) {
		out[i] = in[i] * 2;
	}
}

template<class T>
class Doubler {
	T obj;
	public:
		void vecDouble(int *in, int *out, const size_t n) {
			obj.vecDouble(in, out, n);
		}
};

class HostDoubler {
	public:
		void vecDouble(int *in, int *out, const size_t n) {
			for (size_t i=0; i<n; i++)
				out[i] = in[i] * 2;
		}
};

class DeviceDoubler {
	public:
		static void vecDouble(int *hIn, int *hOut, const size_t n) {
			int *dIn;
			int *dOut;
			hipMalloc((void **)&dIn, n * sizeof(int));
			hipMalloc((void **)&dOut, n * sizeof(int));
			hipMemcpy(dIn, hIn, n * sizeof(int), hipMemcpyHostToDevice);

			kernel_vecDouble<<<1, n>>>(dIn, dOut, n);
			hipDeviceSynchronize();

			hipMemcpy(hOut, dOut, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(dIn);
			hipFree(dOut);
		}
};

static void vecDouble(int *, int *, const size_t);

int main() {
	printf("Hello\n");

	const size_t n = 10;
	int * in = new int[n];
	int * out = new int[n];
	int * answer = new int[n];

	for (size_t i=0; i<n; i++)
		in[i] = rand() % 100;

	Doubler<HostDoubler> host;
	Doubler<DeviceDoubler> device;

	host.vecDouble(in, answer, n);
	device.vecDouble(in, out, n);

	for (size_t i=0; i<n; i++) {
		if (answer[i] != out[i]) {
			printf("error at index %d\n", i);
			break;
		}
	}
	printf("OK\n");

	delete[] in;
	delete[] out;
	delete[] answer;

	return 0;
}
