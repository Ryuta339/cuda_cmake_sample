#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

static void vecDouble(int *, int *, const size_t);

int main() {
	printf("Hello\n");

	const size_t n = 10;
	int * in = new int[n];
	int * out = new int[n];
	int * answer = new int[n];

	for (size_t i=0; i<n; i++)
		in[i] = rand() % 100;
	for (size_t i=0; i<n; i++)
		answer[i] = in[i] * 2;

	vecDouble(in, out, n);

	for (size_t i=0; i<n; i++) {
		if (answer[i] != out[i]) {
			printf("error at index %d\n", i);
			break;
		}
	}
	printf("OK\n");

	delete[] in;
	delete[] out;
	delete[] answer;

	return 0;
}

__global__ void kernel_vecDouble(int *in, int *out, const size_t n) {
	int i = threadIdx.x;
	if (i < n) {
		out[i] = in[i] * 2;
	}
}

static void vecDouble(int *hIn, int *hOut, const size_t n) {
	int *dIn;
	int *dOut;
	hipMalloc((void **)&dIn, n * sizeof(int));
	hipMalloc((void **)&dOut, n * sizeof(int));
	hipMemcpy(dIn, hIn, n * sizeof(int), hipMemcpyHostToDevice);

	kernel_vecDouble<<<1, n>>>(dIn, dOut, n);
	hipDeviceSynchronize();

	hipMemcpy(hOut, dOut, n * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dIn);
	hipFree(dOut);
}
